#include "hip/hip_runtime.h"
#include "nbody.cuh"
#include <iostream>
#include <fstream>
#include <>

int bodies_size_float3 = 0;
int bodies_size_float = 0;
float3 *pos_dev = NULL;
float3 *vel_dev = NULL;
float3 *acc_dev = NULL;
float *m_dev = NULL;
float *r_dev = NULL;

float3 pos[N_SIZE];
float3 vel[N_SIZE];
float3 acc[N_SIZE];
float m[N_SIZE];
float r[N_SIZE];

Camera camera;

GLuint vertexArray;

__device__
int icbrt2(unsigned x) {
   int s;
   unsigned y, b, y2;

   y2 = 0;
   y = 0;
   for (s = 30; s >= 0; s = s - 3) {
      y2 = 4*y2;
      y = 2*y;
      b = (3*(y2 + y) + 1) << s;
      if (x >= b) {
         x = x - b;
         y2 = y2 + 2*y + 1;
         y = y + 1;
      }
   }
   return y;
}

void initBody(int i)
{
	pos[i].x = (-WINDOW_W / 2 + ((float)rand() / (float)(RAND_MAX)) * WINDOW_W) * 0.9;
	pos[i].y = (-WINDOW_H / 2 + ((float)rand() / (float)(RAND_MAX)) * WINDOW_H) * 0.9;
	pos[i].z = (-500 + ((float)rand() / (float)(RAND_MAX)) * 500) * 0.9;

	acc[i].x = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;
	acc[i].y = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;
	acc[i].z = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;

	vel[i].x = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;
	vel[i].y = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;
	vel[i].z = -50 + ((float)rand() / (float)(RAND_MAX)) * 50;

	r[i] = ((float)rand() / (float)(RAND_MAX)) * 3.0;
	m[i] = 4.0 / 3.0 * PI * pow(r[i],3) * DENSITY;
}

void initCUDA()
{


	bodies_size_float3 = N_SIZE * sizeof(float3);
	bodies_size_float = N_SIZE * sizeof(float);

	hipMalloc( (void**)&pos_dev, bodies_size_float3 ); 
	hipMalloc( (void**)&acc_dev, bodies_size_float3 ); 
	hipMalloc( (void**)&vel_dev, bodies_size_float3 ); 
	hipMalloc( (void**)&m_dev, bodies_size_float ); 
	hipMalloc( (void**)&r_dev, bodies_size_float ); 

	for(int i = 0; i < N_SIZE; i++){
		initBody(i);
	}


	hipMemcpy( pos_dev, pos, bodies_size_float3, hipMemcpyHostToDevice );
	hipMemcpy( acc_dev, acc, bodies_size_float3, hipMemcpyHostToDevice );
	hipMemcpy( vel_dev, vel, bodies_size_float3, hipMemcpyHostToDevice );
	hipMemcpy( m_dev, m, bodies_size_float, hipMemcpyHostToDevice );
	hipMemcpy( r_dev, r, bodies_size_float, hipMemcpyHostToDevice );

}

void initGL()
{
    glEnable(GL_CULL_FACE);
	glEnable(GL_POINT_SIZE);

    glEnable(GL_LIGHTING);
    glLightModeli(GL_LIGHT_MODEL_LOCAL_VIEWER, GL_TRUE);

    glEnable(GL_LIGHT0);
    glEnable(GL_COLOR_MATERIAL);

	glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    /*void glOrtho(GLdouble  left,  GLdouble  right,  GLdouble  bottom,  GLdouble  top,  GLdouble  nearVal,  GLdouble  farVal);*/

    gluPerspective (45, (float)WINDOW_W/(float)WINDOW_H, 1, 2000);
   
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

   	gluLookAt(camera.pos.x,camera.pos.y,camera.pos.z, //Camera position
    camera.pos.x+camera.forward.x,camera.pos.y+camera.forward.y,camera.pos.z+camera.forward.z, //Position of the object to look at
    camera.up.x,camera.up.y,camera.up.z); //Camera up direction


	glEnable(GL_DEPTH_TEST);
	glEnable(GL_FOG);
}

// init the program
void init()
{
	initGL();
	initCUDA();
	atexit(deinit);
}

void deinit()
{
	hipFree( pos_dev );
	hipFree( r_dev );
	hipFree( m_dev );
	hipFree( acc_dev );
	hipFree( vel_dev )	;
}

__device__
void updatePosAndVel(float3 pos[], float3 vel[], float3 acc[], float3 cur_a, int self)
{
	float newvx = vel[self].x + (acc[self].x + cur_a.x ) / 2 * TIME_STEP;
	float newvy = vel[self].y + (acc[self].y + cur_a.y ) / 2 * TIME_STEP;
	float newvz = vel[self].z + (acc[self].z + cur_a.z ) / 2 * TIME_STEP;

	//update position
	pos[self].x += newvx * TIME_STEP + acc[self].x * TIME_STEP * TIME_STEP /2;
	pos[self].y += newvy * TIME_STEP + acc[self].y * TIME_STEP * TIME_STEP /2;
	pos[self].z += newvz * TIME_STEP + acc[self].z * TIME_STEP * TIME_STEP /2;

	//update velocity
	vel[self].x = newvx;
	vel[self].y = newvy;
	vel[self].z = newvz; 
}

__device__
void bodyBodyInteraction(float3 &acc, float m[], int self, int other, float3 dist3, float dist_sqr)
{
	float dist_six = dist_sqr * dist_sqr * dist_sqr;
	float dist_cub = sqrtf(dist_six);

	// this is according to the Newton's law of universal gravitaion
	acc.x += (m[other] * dist3.x) / dist_cub;
	acc.y += (m[other] * dist3.y) / dist_cub;
	acc.z += (m[other] * dist3.z) / dist_cub;
}

template <class T>
__device__ void swap(T& first, T& second)
{
	T tmp = first;
	first = second;
	second = tmp;
}

__device__
void mergeBodies(float m[], float3 vel[], float3 acc[], int self, int other)
{
	float newMass = m[self] + m[other];

	// Used perfectly unelastic collision model to caculate the velocity after merging.
	float3 velocity;

	velocity.x = (vel[self].x * m[self] + vel[other].x * m[other]) / newMass;
	velocity.y = (vel[self].y * m[self] + vel[other].y * m[other]) / newMass;
	velocity.z = (vel[self].z * m[self] + vel[other].z * m[other]) / newMass;

	float3 zero_float3 = { 0.0f, 0.0f, 0.0f } ;
	acc[self] = zero_float3;
	acc[other] = zero_float3;

	int biggerIndex = self, smallerIndex = other;

	if (m[biggerIndex] < m[smallerIndex])
	{
		swap(biggerIndex, smallerIndex);
	}

	m[biggerIndex] = newMass;
	vel[biggerIndex] = velocity;
	m[smallerIndex] = 0.0f;
	vel[smallerIndex] = zero_float3;	
}

__global__ 
void nbody(float3* pos, float3* acc, float3* vel, float* m, float* r) 
{
	int idx = blockIdx.x * THREADS + threadIdx.x;

	if (idx >= N_SIZE || m[idx] == 0)
		return;

	float oldMass = m[idx];

	// initiate the acceleration of the next moment 
	float3 cur_acc = { 0.0f, 0.0f, 0.0f };

	// for any two body
	for (int i = 0; i < N_SIZE; i++) {

		if (i != idx && m[i] != 0) {

			if (m[idx] == 0)
				continue;

			float3 dist3; // calculate their distance

			dist3.x = pos[i].x - pos[idx].x;
			dist3.y = pos[i].y - pos[idx].y;
			dist3.z = pos[i].z - pos[idx].z;

			// update the force between two non-empty bodies
			float dist_sqr = dist3.x * dist3.x + dist3.y * dist3.y + dist3.z * dist3.z + SOFT_FACTOR;

			if (sqrt(dist_sqr) > r[idx] + r[i])
				bodyBodyInteraction(cur_acc, m, idx, i, dist3, dist_sqr);
			else
				mergeBodies(m, vel, acc, idx, i);

		}
	}

	// multiplies a Gravitational Constant
	cur_acc.x *= GRAVITATIONAL_CONSTANT;
	cur_acc.y *= GRAVITATIONAL_CONSTANT;
	cur_acc.z *= GRAVITATIONAL_CONSTANT;

	//update the position and velocity
	updatePosAndVel(pos, vel, acc, cur_acc, idx);

	// update the body acceleration
	acc[idx].x = cur_acc.x;
	acc[idx].y = cur_acc.y;
	acc[idx].z = cur_acc.z;

	// if the mass is changed, update the radius
	if (m[idx] != oldMass)
		r[idx] = icbrt2(m[idx] / (DENSITY * 4.0 / 3.0 * PI));
}

int runKernelNBodySimulation()
{
	// Map the buffer to CUDA

	nbody<<<BLOCKS, THREADS>>>(pos_dev, acc_dev, vel_dev, m_dev, r_dev);

	hipMemcpy( pos, pos_dev, bodies_size_float3, hipMemcpyDeviceToHost ); 
	hipMemcpy( m, m_dev, bodies_size_float, hipMemcpyDeviceToHost ); 
	hipMemcpy( r, r_dev, bodies_size_float, hipMemcpyDeviceToHost ); 

	return EXIT_SUCCESS;
}
