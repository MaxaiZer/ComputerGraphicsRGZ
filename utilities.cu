#include "hip/hip_runtime.h"
#include "utilities.cuh"
#include "nbody.cuh"
#include <ctime>
#include <string>

#define ESC_CODE 27

float prevX = WINDOW_W / 2, prevY = WINDOW_H / 2;
bool toggleHelp = true;
bool mouseUp = 0;

extern float4* pos;
extern float4* vel;
extern float4* acc;
extern float* m;
extern float* r;

GLfloat lpos[4] = {-0.3,0.0,200,0}; //������� �����
GLfloat light_specular[4] = {1, 0.6, 1, 0}; //������������� ����������� �����
GLfloat light_diffuse[] = { 1.0, 1.0, 1.0, 0.0 }; //������������� ����������� ����� 
GLfloat light_ambient[] = { 0.2, 0.2, 0.2, 0.0 };  //������������� ����������� �����
GLfloat a;
GLfloat mat_emission[] = {0.8, 0.5, 0.3, 0.0}; // �������� ��������� (��������� �����)
GLfloat mat_specular[] = { 4.0, 0.5, 2.0, 0.0 };  //������������ ��������� �������
GLfloat low_shininess[] = { 50 };
GLfloat fogColor[] = {0.5f, 0.5f, 0.5f, 1};

float fps;
float lastFrameTime = 0;
int frameNumber = 0;

const int FOV = 40;

void resizeCallback(int w, int h)
{
    if (h == 0)
        h = 1;

    float ratio = 1.0 * w / h;

    //����� ������� ���������
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    //��������� ������� ����
    glViewport(0, 0, w, h);

    //��������� �����������
    gluPerspective(45, ratio, 1, 1000);
    glMatrixMode(GL_MODELVIEW);
}

void handleKeyboard(unsigned char key, int x, int y)
{
    if (key == ESC_CODE)
        exit(0);

    float vel = 5.0;
    float rightX, rightY, rightZ;
    cross(camera.forward.x, camera.forward.y, camera.forward.z, camera.up.x, camera.up.y, camera.up.z, rightX, rightY, rightZ);
    float sizeRight = sqrtf(rightX * rightX + rightY * rightY + rightZ * rightZ);
    rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;

    int coef1 = 0;
    int coef2 = 0;

    switch (key)
    {
    case 'w':
        coef1 = 1; // �����
        break;
    case 's':
        coef1 = -1; // �����
        break;
    case 'a':
        coef2 = -1; // �����
        break;
    case 'd':
        coef2 = 1; // ������
        break;
    }

    if (coef1 != 0)
    {
        camera.pos.x += camera.forward.x * vel * coef1;
        camera.pos.y += camera.forward.y * vel * coef1;
        camera.pos.z += camera.forward.z * vel * coef1;
    }

    if (coef2 != 0)
    {
        camera.pos.x += rightX * vel * coef2;
        camera.pos.y += rightY * vel * coef2;
        camera.pos.z += rightZ * vel * coef2;
    }

    if (key == 'h') // show or hide help
    {
        toggleHelp = !toggleHelp;
    }
}

void saveMousePos(int x, int y)
{
    prevX = x, prevY = y;
}

void handleMouse(int x, int y)
{
    float velx = (float(x - prevX) / WINDOW_W);
    float vely = (float(y - prevY) / WINDOW_H);
    prevX = x;
    prevY = y;
    camera.phi += -velx * PI * 0.9;
    camera.theta += -vely * PI * 0.9;

    float rightX, rightY, rightZ;
    rightX = sinf(camera.phi - PI / 2.0f);
    rightY = 0;
    rightZ = cosf(camera.phi - PI / 2.0f);
    float sizeRight = sqrtf(rightX * rightX + rightY * rightY + rightZ * rightZ);
    rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;


    float cosTheta = cosf(camera.theta);
    camera.forward.x = cosTheta * sinf(camera.phi);
    camera.forward.y = sinf(camera.theta);
    camera.forward.z = cosTheta * cosf(camera.phi);

    float sizeForward = sqrtf(camera.forward.x * camera.forward.x + camera.forward.y * camera.forward.y + camera.forward.z * camera.forward.z);
    camera.forward.x /= sizeForward; camera.forward.y /= sizeForward; camera.forward.z /= sizeForward;

    float newUpX, newUpY, newUpZ;

    cross(rightX, rightY, rightZ, camera.forward.x, camera.forward.y, camera.forward.z, newUpX, newUpY, newUpZ);
    float sizeUp = sqrtf(newUpX * newUpX + newUpY * newUpY + newUpZ * newUpZ);
    camera.up.x = newUpX / sizeUp; camera.up.y = newUpY / sizeUp; camera.up.z = newUpZ / sizeUp;
}

float getVectorsAngle(float4 v1, float4 v2)
{
    float cos = (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z) /
        ( sqrtf(v1.x * v1.x + v1.y * v1.y + v1.z * v1.z) * 
            sqrtf(v2.x * v2.x + v2.y * v2.y + v2.z * v2.z) );
    return acos(cos) * 180.0 / PI;
}

void cross(float x1, float y1, float z1, float x2, float y2, float z2,float& rightX, float& rightY, float& rightZ)
{
    rightX = y1*z2 - z1*y2;
    rightY = x1*z2 - x1*z2;
    rightZ = x1*y2 - y1*x1;
}

void setLights()
{
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightfv(GL_LIGHT0, GL_POSITION, lpos);

    //�����
    glFogfv(GL_FOG_COLOR, fogColor);
    glFogi(GL_FOG_MODE, GL_LINEAR);
    glFogf(GL_FOG_START, 10.0f);
    glFogf(GL_FOG_END, 1000.0f);
}

void calculateFPS()
{
    float currentTime = clock();// / 1000.0;
    fps = (float)1000 / (currentTime - lastFrameTime);
    lastFrameTime = currentTime;
    frameNumber++;
}

void drawScene()
{
    calculateFPS();

    glClearColor(0.1f,0.1f,0.1f,0.1f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(camera.pos.x,camera.pos.y,camera.pos.z, //Camera position
              camera.pos.x+camera.forward.x,camera.pos.y+camera.forward.y,camera.pos.z+camera.forward.z, //Position of the object to look at
              camera.up.x,camera.up.y,camera.up.z); //Camera up direction
    setLights();

	runKernelNBodySimulation();

    if (frameNumber % 40 == 0)
    {
        printf("FPS: %.3f\n", fps);
    }

    glColor3f(0.5f, 0.5f, 0.3f);
    
    for(int i = 0; i < BODIES; i ++)
    {
        if (m[i] == 0)
            continue;

        float4 bodyVector = { pos[i].x - camera.pos.x, pos[i].y - camera.pos.y, pos[i].z - camera.pos.z };
        if (getVectorsAngle(camera.forward, bodyVector) > FOV)
            continue;

		glPushMatrix();
		glTranslatef(pos[i].x, pos[i].y, pos[i].z);
		glutSolidSphere(r[i], 10, 10); // draw sphere
		glPopMatrix();          
    }

    glutSwapBuffers();

}