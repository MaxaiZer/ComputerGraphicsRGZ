#include "hip/hip_runtime.h"
#include "util.cuh"
#include "nbody.cuh"
#include <ctime>
#include <string>

#define ESC_CODE 27

float prevX = WINDOW_W / 2, prevY = WINDOW_H / 2;
bool toggleHelp = true;
bool mouseUp = 0;

extern float4 pos[N_SIZE];
extern float4 vel[N_SIZE];
extern float4 acc[N_SIZE];
extern float m[N_SIZE];
extern float r[N_SIZE];

GLfloat lpos[4] = {-0.3,0.0,200,0}; //Positioned light
GLfloat light_specular[4] = {1, 0.6, 1, 0}; //specular light intensity (color)
GLfloat light_diffuse[] = { 1.0, 1.0, 1.0, 0.0 };//diffuse light intensity (color)
GLfloat light_ambient[] = { 0.2, 0.2, 0.2, 0.0 }; //ambient light intensity (color)
GLfloat a;
GLfloat mat_emission[] = {0.8, 0.5, 0.3, 0.0}; //object material preperty emission of light
GLfloat mat_specular[] = { 4.0, 0.5, 2.0, 0.0 }; //object material specularity
GLfloat low_shininess[] = { 50 };
GLfloat fogColor[] = {0.5f, 0.5f, 0.5f, 1};

float fps;
float lastFrameTime = 0;

const int FOV = 40;

//void timerFunc(int value)
//{
//    glutPostRedisplay();
//}

void resizeCallback(int w, int h)
{
    // Prevent a divide by zero, when window is too short
    // (you cant make a window of zero width).
    if (h == 0)
        h = 1;

    float ratio = 1.0 * w / h;

    // Reset the coordinate system before modifying
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();

    // Set the viewport to be the entire window
    glViewport(0, 0, w, h);

    // Set the correct perspective.
    gluPerspective(45, ratio, 1, 1000);
    glMatrixMode(GL_MODELVIEW);

}

void keyboardFunc(unsigned char key, int x, int y)
{
    if (key == ESC_CODE)
        exit(0);

    float vel = 5.0;
    float rightX, rightY, rightZ;
    cross(camera.forward.x, camera.forward.y, camera.forward.z, camera.up.x, camera.up.y, camera.up.z, rightX, rightY, rightZ);
    float sizeRight = sqrtf(rightX * rightX + rightY * rightY + rightZ * rightZ);
    rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;

    int coef1 = 0;
    int coef2 = 0;

    switch (key)
    {
    case 'w':
        coef1 = 1; // move forward
        break;
    case 's':
        coef1 = -1; // move backward
        break;
    case 'a':
        coef2 = -1; // move left
        break;
    case 'd':
        coef2 = 1; // move right
        break;
    }

    if (coef1 != 0)
    {
        camera.pos.x += camera.forward.x * vel * coef1;
        camera.pos.y += camera.forward.y * vel * coef1;
        camera.pos.z += camera.forward.z * vel * coef1;
    }

    if (coef2 != 0)
    {
        camera.pos.x += rightX * vel * coef2;
        camera.pos.y += rightY * vel * coef2;
        camera.pos.z += rightZ * vel * coef2;
    }

    if (key == 'h') // show or hide help
    {
        toggleHelp = !toggleHelp;
    }
}

void PassiveMouseMotion(int x, int y)
{
    prevX = x, prevY = y;
}

// call back function triggered by mouse
void mouseCallback(int x, int y)
{
    float velx = (float(x - prevX) / WINDOW_W);
    float vely = (float(y - prevY) / WINDOW_H);
    prevX = x;
    prevY = y;
    camera.phi += -velx * PI * 0.9;
    camera.theta += -vely * PI * 0.9;

    float rightX, rightY, rightZ;
    rightX = sinf(camera.phi - PI / 2.0f);
    rightY = 0;
    rightZ = cosf(camera.phi - PI / 2.0f);
    float sizeRight = sqrtf(rightX * rightX + rightY * rightY + rightZ * rightZ);
    rightX /= sizeRight; rightY /= sizeRight; rightZ /= sizeRight;


    camera.forward.x = cosf(camera.theta) * sinf(camera.phi);
    camera.forward.y = sinf(camera.theta);
    camera.forward.z = cosf(camera.theta) * cosf(camera.phi);

    float sizeForward = sqrtf(camera.forward.x * camera.forward.x + camera.forward.y * camera.forward.y + camera.forward.z * camera.forward.z);
    camera.forward.x /= sizeForward; camera.forward.y /= sizeForward; camera.forward.z /= sizeForward;

    float newUpX, newUpY, newUpZ;

    cross(rightX, rightY, rightZ, camera.forward.x, camera.forward.y, camera.forward.z, newUpX, newUpY, newUpZ);
    float sizeUp = sqrtf(newUpX * newUpX + newUpY * newUpY + newUpZ * newUpZ);
    camera.up.x = newUpX / sizeUp; camera.up.y = newUpY / sizeUp; camera.up.z = newUpZ / sizeUp;
}

float getVectorsAngle(float4 v1, float4 v2)
{
    float cos = (v1.x * v2.x + v1.y * v2.y + v1.z * v2.z) /
        ( sqrtf(v1.x * v1.x + v1.y * v1.y + v1.z * v1.z) * 
            sqrtf(v2.x * v2.x + v2.y * v2.y + v2.z * v2.z) );
    return acos(cos) * 180.0 / PI;
}

void cross(float x1, float y1, float z1, float x2, float y2, float z2,float& rightX, float& rightY, float& rightZ)
{
    rightX = y1*z2 - z1*y2;
    rightY = x1*z2 - x1*z2;
    rightZ = x1*y2 - y1*x1;
}

void drawText(std::string text, float x, float y)
{
    glMatrixMode(GL_MODELVIEW);
    glPushMatrix();

    glColor3f(1.0f, 0.0f, 0.0f);//needs to be called before RasterPos
    glRasterPos2f(x, y);
    
    void * font = GLUT_BITMAP_TIMES_ROMAN_24;

    for (std::string::iterator i = text.begin(); i != text.end(); ++i)
    {
        char c = *i;
        glutBitmapCharacter(font, c);
    }
    glPopMatrix();
}

void setLights()
{
    glMaterialfv(GL_FRONT, GL_EMISSION, mat_emission);
    glMaterialfv(GL_FRONT, GL_SPECULAR, mat_specular);
    glMaterialfv(GL_FRONT, GL_SHININESS, low_shininess);
    glLightfv(GL_LIGHT0, GL_SPECULAR, light_specular);
    glLightfv(GL_LIGHT0, GL_POSITION, lpos);

    //Adding fog
    glFogfv(GL_FOG_COLOR, fogColor);
    glFogi(GL_FOG_MODE, GL_LINEAR);
    glFogf(GL_FOG_START, 10.0f);
    glFogf(GL_FOG_END, 1000.0f);
}

void calculateFPS()
{
    float currentTime = clock();// / 1000.0;
    fps = (float)1000 / (currentTime - lastFrameTime);
    lastFrameTime = currentTime;
}

void draw2()
{
    calculateFPS();

    glClearColor(0.1f,0.1f,0.1f,0.1f);
    glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
    

    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
    gluLookAt(camera.pos.x,camera.pos.y,camera.pos.z, //Camera position
              camera.pos.x+camera.forward.x,camera.pos.y+camera.forward.y,camera.pos.z+camera.forward.z, //Position of the object to look at
              camera.up.x,camera.up.y,camera.up.z); //Camera up direction
    setLights();

	runKernelNBodySimulation();

    if (toggleHelp)
    {
        drawText("USAGE INFO", 50, 60);
        drawText("Use keys w, a, s, d to move", 50, 50);
        drawText("Hold the left button on the mouse to look around", 50, 40);
        drawText("Press h to show/hide this help info", 50, 30);
    }

    drawText("FPS: " + std::to_string(fps), camera.pos.x + camera.forward.x, camera.pos.y + camera.forward.y);

    glColor3f(0.5f, 0.5f, 0.3f);
    
    for(int i = 0; i < N_SIZE; i ++)
    {
        if (m[i] == 0)
            continue;

        float4 bodyVector = { pos[i].x - camera.pos.x, pos[i].y - camera.pos.y, pos[i].z - camera.pos.z };
        if (getVectorsAngle(camera.forward, bodyVector) > FOV)
            continue;

		glPushMatrix();
		glTranslatef(pos[i].x, pos[i].y, pos[i].z);
		glutSolidSphere(r[i], 10, 10); // draw sphere
		glPopMatrix();          
    }

    glutSwapBuffers();

}